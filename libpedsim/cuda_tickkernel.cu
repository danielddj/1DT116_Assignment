#include "hip/hip_runtime.h"
#include "ped_model.h"
#include "ped_waypoint.h"
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>

#define HEATMAP_WIDTH 160 * 5
#define HEATMAP_HEIGHT 120 * 5
#define HEATMAP_SKIP 5

bool firstRun = true;

struct Twaypoint
{
  float x;
  float y;
  float r; // threshold radius for "arrival"
};

#define CUDA_CHECK(call)                                                                                           \
  {                                                                                                                \
    hipError_t err = call;                                                                                        \
    if (err != hipSuccess)                                                                                        \
    {                                                                                                              \
      fprintf(stderr, "CUDA Error: %s (err_num=%d) at %s:%d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
      exit(err);                                                                                                   \
    }                                                                                                              \
  }

__device__ Twaypoint getNextDestination(const float *agentX, const float *agentY, const int *agentWaypoints,
                                        size_t agentWaypointsPitch, const float *waypointX, const float *waypointY,
                                        const float *waypointR, int numAgents, int *waypointIndex, int numWaypoints)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  // Default destination.
  Twaypoint dest = {0.0f, 0.0f, 0.0f};

  if (id >= numAgents)
    return dest;

  // Get current waypoint index for this agent.
  int wpIndex = waypointIndex[id];

  // Validate wpIndex before use.
  if (wpIndex < 0 || wpIndex >= numWaypoints)
  {
    wpIndex = 0;
    waypointIndex[id] = 0; // update agent's index to a valid value.
  }

  // Get pointer to the agent's waypoint list.
  const int *rowPtr = (const int *)((const char *)agentWaypoints + id * agentWaypointsPitch);

  // Read the waypoint ID for the current waypoint.
  int wpID = rowPtr[wpIndex];

  // Read destination values from the global waypoint arrays.
  float destX = waypointX[wpID - 1];
  float destY = waypointY[wpID - 1];
  float destR = waypointR[wpID - 1];

  // Compute distance between agent's current position and destination.
  float diffX = destX - agentX[id];
  float diffY = destY - agentY[id];
  float dist = sqrtf(diffX * diffX + diffY * diffY);

  // Check if the agent has reached its current destination.
  bool reached = (dist < destR);
  if (reached)
  {
    // Advance to the next waypoint in a circular manner.
    wpIndex = (wpIndex + 1) % numWaypoints;
    waypointIndex[id] = wpIndex;

    // Update the waypoint ID to the new one.
    wpID = rowPtr[wpIndex];

    destX = waypointX[wpID];
    destY = waypointY[wpID];
    destR = waypointR[wpID];
  }

  dest.x = destX;
  dest.y = destY;
  dest.r = destR;

  // Set the destination using the (possibly updated) waypoint ID.

  return dest;
}

__device__ void computeNextDesiredPosition(float *d_bufferXSim, float *d_bufferYSim, float *agentDesX, float *agentDesY, int *agentWaypoints,
                                           size_t agentWaypointsPitch, float *waypointX, float *waypointY,
                                           float *waypointR, int numAgents, int *waypointIndex, int numWaypoints)
{
  Twaypoint dest = getNextDestination(d_bufferXSim, d_bufferYSim, agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents,
                                      waypointIndex, numWaypoints);

  int agentId = blockIdx.x * blockDim.x + threadIdx.x;
  if (agentId < numAgents)
  {
    float diffX = dest.x - d_bufferXSim[agentId];
    float diffY = dest.y - d_bufferYSim[agentId];
    float length = sqrt(diffX * diffX + diffY * diffY);

    d_bufferXSim[agentId] = d_bufferXSim[agentId] + diffX / length;
    d_bufferYSim[agentId] = d_bufferYSim[agentId] + diffY / length;
  }
}

__global__ void cudaTickKernel(float *d_bufferXSim, float *d_bufferX2Transfer, float *d_bufferYSim,
                               float *d_bufferY2Transfer, float *agentDesX, float *agentDesY,
                               int *agentWaypoints, size_t agentWaypointsPitch, float *waypointX,
                               float *waypointY, float *waypointR, int numAgents, int *waypointIndex,
                               int numWaypoints)
{
  computeNextDesiredPosition(d_bufferXSim, d_bufferYSim, agentDesX, agentDesY, agentWaypoints,
                             agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents, waypointIndex,
                             numWaypoints);

  int agentId = blockIdx.x * blockDim.x + threadIdx.x;

  if (agentId < numAgents)
  {
    d_bufferX2Transfer[agentId] = d_bufferXSim[agentId];
    d_bufferY2Transfer[agentId] = d_bufferYSim[agentId];
  }
}

void serializeDataCuda(const float *h_exportBufferX, const float *h_exportBufferY, const Ped::Model &model,
                       std::ofstream &file)
{
  const std::vector<Ped::Tagent *> &agents = model.getAgents();
  size_t num_agents = agents.size();

  file.write(reinterpret_cast<const char *>(&num_agents), sizeof(num_agents));

  for (size_t i = 0; i < num_agents; i++)
  {

    int16_t x = static_cast<int16_t>(h_exportBufferX[i]);
    int16_t y = static_cast<int16_t>(h_exportBufferY[i]);

    file.write(reinterpret_cast<const char *>(&x), sizeof(x));
    file.write(reinterpret_cast<const char *>(&y), sizeof(y));
  }

  size_t heatmap_elements = model.getHeatmapSize();
  int16_t height = HEATMAP_HEIGHT;
  int16_t width = HEATMAP_WIDTH;
  const int *const *heatmap = model.getHeatmap();

  unsigned long heatmap_start = 0xFFFF0000FFFF0000;
  file.write(reinterpret_cast<const char *>(&heatmap_start), sizeof(heatmap_start));
  printf("heatmap_start: %ld\n", sizeof(heatmap_start));

  for (int i = 0; i < height; i++)
  {
    for (int j = 0; j < width; j++)
    {
      int ARGBvalue = heatmap[i][j];
      int8_t Avalue = (ARGBvalue >> 24) & ((1 << 8) - 1);
      file.write(reinterpret_cast<const char *>(&Avalue), sizeof(Avalue));
    }
  }

  file.flush();
}

namespace Ped
{
  size_t Ped::Model::tick_cuda(size_t ticks, float *d_bufferX1, float *d_bufferX2, float *d_bufferY1, float *d_bufferY2, float *agentDesX, float *agentDesY,
                               float *waypointX, float *waypointY, float *waypointR, int *agentWaypoints,
                               size_t agentWaypointsPitch, int *waypointIndex, bool serialize, std::ofstream *file)
  {
    // Calculate the number of blocks needed
    size_t tickCount;

    int threadsPerBlock = 256;
    int numBlocks = (X.size() + threadsPerBlock - 1) / 256;

    bool useBuffer1ForSim = true;

    int numAgents = X.size();
    int numWaypoints = X_WP.size();

    size_t size_agent = numAgents * sizeof(float);

    float *h_exportBufferX = nullptr, *h_exportBufferY = nullptr;
    hipHostMalloc((void **)&h_exportBufferX, size_agent);
    hipHostMalloc((void **)&h_exportBufferY, size_agent);

    hipStream_t stream;
    hipStreamCreate(&stream);
    bool firstFrame = true;

    for (size_t i = 0; i < ticks; i++)
    {
      // Launch the kernel with 256 threads per block.
      if (useBuffer1ForSim)
      {
        cudaTickKernel<<<numBlocks, threadsPerBlock>>>(
            d_bufferX1, d_bufferX2, d_bufferY1, d_bufferY2, agentDesX, agentDesY,
            agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR,
            numAgents, waypointIndex, numWaypoints);
      }
      else
      {
        cudaTickKernel<<<numBlocks, threadsPerBlock>>>(
            d_bufferX2, d_bufferX1, d_bufferY2, d_bufferY1, agentDesX, agentDesY,
            agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR,
            numAgents, waypointIndex, numWaypoints);
      }

      // Copy device simulation data to host export buffers.

      if (firstFrame)
      {
        hipMemcpy(h_exportBufferX, (useBuffer1ForSim ? d_bufferX1 : d_bufferX2), size_agent, hipMemcpyDeviceToHost);
        hipMemcpy(h_exportBufferY, (useBuffer1ForSim ? d_bufferY1 : d_bufferY2), size_agent, hipMemcpyDeviceToHost);
        firstFrame = false;
      }
      if (useBuffer1ForSim)
      {
        hipMemcpyAsync(h_exportBufferX, d_bufferX1, size_agent, hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_exportBufferY, d_bufferY1, size_agent, hipMemcpyDeviceToHost, stream);
      }
      else
      {
        hipMemcpyAsync(h_exportBufferX, d_bufferX2, size_agent, hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_exportBufferY, d_bufferY2, size_agent, hipMemcpyDeviceToHost, stream);
      }

      // Now the host export buffers are ready.
      if (serialize)
      {
        serializeDataCuda(h_exportBufferX, h_exportBufferY, *this, *file);
      }

      tickCount++;
      useBuffer1ForSim = !useBuffer1ForSim;
    }

    hipHostFree(h_exportBufferX);
    hipHostFree(h_exportBufferY);

    return tickCount;
  }

  size_t Ped::Model::start_cuda(size_t maxSteps, bool serialize, std::ofstream *file)
  {

    float *agentStartX, *agentStartY, *agentDesX, *agentDesY, *waypointX, *waypointY, *waypointR;
    int *agentWaypoints, *waypointIndex;

    int numAgents = X.size();
    int numWaypoints = X_WP.size();

    size_t size_agent = numAgents * sizeof(float);
    size_t size_waypoint = numWaypoints * sizeof(int);
    size_t pitch;

    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    CUDA_CHECK(hipMallocAsync(&agentStartX, size_agent, stream1));
    CUDA_CHECK(hipMallocAsync(&agentStartY, size_agent, stream2));
    CUDA_CHECK(hipMallocAsync(&agentDesX, size_agent, stream3))
    CUDA_CHECK(hipMallocAsync(&agentDesY, size_agent, stream4));
    CUDA_CHECK(hipMallocPitch(&agentWaypoints, &pitch, size_waypoint, size_agent));
    CUDA_CHECK(hipMallocAsync(&waypointX, size_agent, stream1));
    CUDA_CHECK(hipMallocAsync(&waypointY, size_agent, stream2));
    CUDA_CHECK(hipMallocAsync(&waypointR, size_agent, stream3));
    CUDA_CHECK(hipMallocAsync(&waypointIndex, size_agent, stream4));

    float *d_bufferX1 = nullptr, *d_bufferX2 = nullptr, *d_bufferY1 = nullptr, *d_bufferY2 = nullptr;
    hipMallocAsync((void **)&d_bufferX1, size_agent, stream1);
    hipMallocAsync((void **)&d_bufferX2, size_agent, stream2);
    hipMallocAsync((void **)&d_bufferY1, size_agent, stream3);
    hipMallocAsync((void **)&d_bufferY2, size_agent, stream4);

    CUDA_CHECK(hipMemcpy(d_bufferX1, X.data(), size_agent, hipMemcpyHostToDevice));
    hipMemset(d_bufferX2, 0, size_agent);
    CUDA_CHECK(hipMemcpy(d_bufferY1, Y.data(), size_agent, hipMemcpyHostToDevice));
    hipMemset(d_bufferY2, 0, size_agent);

    CUDA_CHECK(hipMemset(agentDesX, 0, size_agent));
    CUDA_CHECK(hipMemset(agentDesY, 0, size_agent));

    int *agentWaypointsTmp = new int[numAgents * numWaypoints];

    for (size_t i = 0; i < numAgents; i++)
    {
      // Insert the new waypoint at the front and push the rest back
      for (size_t j = numWaypoints - 1; j > 0; j--)
      {
        agentWaypointsTmp[i * numWaypoints + j] = agents.at(i)->getDestination().at(j - 1)->getid();
      }
      // Insert the first waypoint at the front
      agentWaypointsTmp[i * numWaypoints] = agents.at(i)->getDestination().at(numWaypoints - 1)->getid();
    }

    CUDA_CHECK(hipMemcpy2DAsync(agentWaypoints, pitch, agentWaypointsTmp, size_waypoint, size_waypoint, numAgents,
                                 hipMemcpyHostToDevice, stream1));

    CUDA_CHECK(hipMemcpyAsync(waypointX, X_WP.data(), size_waypoint, hipMemcpyHostToDevice, stream2));
    CUDA_CHECK(hipMemcpyAsync(waypointY, Y_WP.data(), size_waypoint, hipMemcpyHostToDevice, stream3));
    CUDA_CHECK(hipMemcpyAsync(waypointR, R_WP.data(), size_waypoint, hipMemcpyHostToDevice, stream4));

    int *waypointIndexTmp = new int[numAgents];

    for (size_t i = 0; i < numAgents; i++)
    {
      waypointIndexTmp[i] = agents.at(i)->getDestination().at(0)->getid();
    }

    CUDA_CHECK(hipMemcpyAsync(waypointIndex, waypointIndexTmp, size_agent, hipMemcpyHostToDevice, stream1));

    if (serialize && !file)
    {
      std::runtime_error("File needs to be open!");
    }

    size_t tickCount = tick_cuda(maxSteps, d_bufferX1, d_bufferX2, d_bufferY1, d_bufferY2, agentDesX, agentDesY, waypointX, waypointY, waypointR, agentWaypoints, pitch,
                                 waypointIndex, serialize, file);

    // Free allocated memory
    hipFree(agentStartX);
    hipFree(agentStartY);
    hipFree(agentDesX);
    hipFree(agentDesY);

    hipFree(agentWaypoints);
    hipFree(waypointX);
    hipFree(waypointY);
    hipFree(waypointR);
    hipFree(waypointIndex);

    hipFree(d_bufferX1);
    hipFree(d_bufferX2);
    hipFree(d_bufferY1);
    hipFree(d_bufferY2);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);

    return tickCount;
  }

  void Ped::Model::warmup()
  {
    hipFree(0);
  }
} // namespace Ped
