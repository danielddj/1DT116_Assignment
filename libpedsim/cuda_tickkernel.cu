#include "hip/hip_runtime.h"
#include "ped_model.h"
#include "ped_waypoint.h"
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>

#define HEATMAP_WIDTH 160 * 5
#define HEATMAP_HEIGHT 120 * 5
#define HEATMAP_SKIP 5

bool firstRun = true;

struct Twaypoint
{
  float x;
  float y;
  float r; // threshold radius for "arrival"
};

#define CUDA_CHECK(call)                                                                                           \
  {                                                                                                                \
    hipError_t err = call;                                                                                        \
    if (err != hipSuccess)                                                                                        \
    {                                                                                                              \
      fprintf(stderr, "CUDA Error: %s (err_num=%d) at %s:%d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
      exit(err);                                                                                                   \
    }                                                                                                              \
  }

__device__ Twaypoint getNextDestination(const float *agentX, const float *agentY, const int *agentWaypoints,
                                        size_t agentWaypointsPitch, const float *waypointX, const float *waypointY,
                                        const float *waypointR, int numAgents, int *waypointIndex, int numWaypoints)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  // Default destination.
  Twaypoint dest = {0.0f, 0.0f, 0.0f};

  if (id >= numAgents)
    return dest;

  // Get current waypoint index for this agent.
  int wpIndex = waypointIndex[id];

  // Validate wpIndex before use.
  if (wpIndex < 0 || wpIndex >= numWaypoints)
  {
    wpIndex = 0;
    waypointIndex[id] = 0; // update agent's index to a valid value.
  }

  // Get pointer to the agent's waypoint list.
  const int *rowPtr = (const int *)((const char *)agentWaypoints + id * agentWaypointsPitch);

  // Read the waypoint ID for the current waypoint.
  int wpID = rowPtr[wpIndex];

  // Read destination values from the global waypoint arrays.
  float destX = waypointX[wpID - 1];
  float destY = waypointY[wpID - 1];
  float destR = waypointR[wpID - 1];

  // Compute distance between agent's current position and destination.
  float diffX = destX - agentX[id];
  float diffY = destY - agentY[id];
  float dist = sqrtf(diffX * diffX + diffY * diffY);

  // Check if the agent has reached its current destination.
  bool reached = (dist < destR);
  if (reached)
  {
    // Advance to the next waypoint in a circular manner.
    wpIndex = (wpIndex + 1) % numWaypoints;
    waypointIndex[id] = wpIndex;

    // Update the waypoint ID to the new one.
    wpID = rowPtr[wpIndex];
  }

  // Set the destination using the (possibly updated) waypoint ID.
  dest.x = destX;
  dest.y = destY;
  dest.r = destR;

  return dest;
}

__device__ void computeNextDesiredPosition(float *d_bufferXSim, float *d_bufferYSim, float *agentX, float *agentY,
                                           float *agentDesX, float *agentDesY, int *agentWaypoints,
                                           size_t agentWaypointsPitch, float *waypointX, float *waypointY,
                                           float *waypointR, int numAgents, int *waypointIndex, int numWaypoints)
{
  getNextDestination(agentX, agentY, agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents,
                     waypointIndex, numWaypoints);

  int agentId = blockIdx.x * blockDim.x + threadIdx.x;
  if (agentId < numAgents)
  {
    float diffX = waypointX[agentWaypoints[agentId]] - agentX[agentId];
    float diffY = waypointY[agentWaypoints[agentId]] - agentY[agentId];
    float length = sqrt(diffX * diffX + diffY * diffY);

    d_bufferXSim[agentId] = agentX[agentId] + diffX / length;
    d_bufferYSim[agentId] = agentY[agentId] + diffY / length;
  }
}

//-----------------------------------------------------------------------------
// CUDA kernel to update agent positions.
// For each agent i, we do roughly the same as in sequential_tick():
//   diff = destination - current position
//   len = sqrt(diff.x*diff.x + diff.y*diff.y)
//   new position = current position + (diff/len)  (rounded)
// Then we store the new positions in desiredX/Y and update X/Y.
//-----------------------------------------------------------------------------
__global__ void cudaTickKernel(float *d_bufferX1Sim, float *d_bufferX2Transfer, float *d_bufferYSim,
                               float *d_bufferY2Transfer, float *agentX, float *agentY, float *agentDesX,
                               float *agentDesY, int *agentWaypoints, size_t agentWaypointsPitch, float *waypointX,
                               float *waypointY, float *waypointR, int numAgents, int *waypointIndex,
                               int numWaypoints)
{
  // Compute the new desired positions
  computeNextDesiredPosition(d_bufferX1Sim, d_bufferYSim, agentX, agentY, agentDesX, agentDesY, agentWaypoints,
                             agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents, waypointIndex,
                             numWaypoints);

  // Wait for all threads to finish
  __syncthreads();

  // Update the agent positions
  for (int i = 0; i < numAgents; i++)
  {
    agentX[i] = d_bufferX1Sim[i];
    agentY[i] = d_bufferYSim[i];
  }

  // Wait for all threads to finish
  __syncthreads();
}

void serializeDataCuda(const float *h_exportBufferX, const float *h_exportBufferY, const Ped::Model &model,
                       std::ofstream &file)
{
  const std::vector<Ped::Tagent *> &agents = model.getAgents();
  size_t num_agents = agents.size();

  file.write(reinterpret_cast<const char *>(&num_agents), sizeof(num_agents));

  for (size_t i = 0; i < num_agents; i++)
  {

    int16_t x = static_cast<int16_t>(h_exportBufferX[i]);
    int16_t y = static_cast<int16_t>(h_exportBufferY[i]);

    file.write(reinterpret_cast<const char *>(&x), sizeof(x));
    file.write(reinterpret_cast<const char *>(&y), sizeof(y));
  }

  size_t heatmap_elements = model.getHeatmapSize();
  int16_t height = HEATMAP_HEIGHT;
  int16_t width = HEATMAP_WIDTH;
  const int *const *heatmap = model.getHeatmap();

  unsigned long heatmap_start = 0xFFFF0000FFFF0000;
  file.write(reinterpret_cast<const char *>(&heatmap_start), sizeof(heatmap_start));
  printf("heatmap_start: %ld\n", sizeof(heatmap_start));

  for (int i = 0; i < height; i++)
  {
    for (int j = 0; j < width; j++)
    {
      int ARGBvalue = heatmap[i][j];
      int8_t Avalue = (ARGBvalue >> 24) & ((1 << 8) - 1);
      file.write(reinterpret_cast<const char *>(&Avalue), sizeof(Avalue));
    }
  }

  file.flush();
}

namespace Ped
{
  void Ped::Model::tick_cuda(size_t ticks, float *agentStartX, float *agentStartY, float *agentDesX, float *agentDesY,
                             float *waypointX, float *waypointY, float *waypointR, int *agentWaypoints,
                             size_t agentWaypointsPitch, int *waypointIndex)
  {
    // Calculate the number of blocks needed
    int threadsPerBlock = 256;
    int numBlocks = (X.size() + threadsPerBlock - 1) / 256;

    bool useBuffer1ForSim = true;

    int numAgents = X.size();
    int numWaypoints = X_WP.size();

    size_t size_agent = numAgents * sizeof(float);

    float *d_bufferX1 = nullptr, *d_bufferX2 = nullptr, *d_bufferY1 = nullptr, *d_bufferY2 = nullptr;
    hipMalloc((void **)&d_bufferX1, size_agent);
    hipMalloc((void **)&d_bufferX2, size_agent);
    hipMalloc((void **)&d_bufferY1, size_agent);
    hipMalloc((void **)&d_bufferY2, size_agent);

    hipMemset(d_bufferX1, 0, size_agent);
    hipMemset(d_bufferX2, 0, size_agent);
    hipMemset(d_bufferY1, 0, size_agent);
    hipMemset(d_bufferY2, 0, size_agent);

    float *h_exportBufferX = nullptr, *h_exportBufferY = nullptr;
    hipHostMalloc((void **)&h_exportBufferX, size_agent);
    hipHostMalloc((void **)&h_exportBufferY, size_agent);

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (size_t i = 0; i < ticks; i++)
    {
      // Launch the kernel with 256 threads per block
      if (useBuffer1ForSim)
      {
        cudaTickKernel<<<numBlocks, threadsPerBlock>>>(
            d_bufferX1, d_bufferX2, d_bufferY1, d_bufferY2, agentStartX, agentStartY, agentDesX, agentDesY,
            agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents, waypointIndex, numWaypoints);
      }
      else
      {
        cudaTickKernel<<<numBlocks, threadsPerBlock>>>(
            d_bufferX2, d_bufferX1, d_bufferY2, d_bufferY1, agentStartX, agentStartY, agentDesX, agentDesY,
            agentWaypoints, agentWaypointsPitch, waypointX, waypointY, waypointR, numAgents, waypointIndex, numWaypoints);
      }

      if (useBuffer1ForSim)
      {
        hipMemcpyAsync(h_exportBufferX, d_bufferX2, size_agent, hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_exportBufferY, d_bufferY2, size_agent, hipMemcpyDeviceToHost, stream);
      }
      else
      {
        hipMemcpyAsync(h_exportBufferX, d_bufferX1, size_agent, hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(h_exportBufferY, d_bufferY1, size_agent, hipMemcpyDeviceToHost, stream);
      }

      serializeDataCuda(h_exportBufferX, h_exportBufferY, *this, file);

      useBuffer1ForSim = !useBuffer1ForSim;
    }
  }

  void Ped::Model::start_cuda()
  {

    float *agentStartX, *agentStartY, *agentDesX, *agentDesY, *waypointX, *waypointY, *waypointR;
    int *agentWaypoints, *waypointIndex;

    int numAgents = X.size();
    int numWaypoints = X_WP.size();

    size_t size_agent = numAgents * sizeof(float);
    size_t size_waypoint = numWaypoints * sizeof(int);
    size_t pitch;

    CUDA_CHECK(hipMalloc(&agentStartX, size_agent));
    CUDA_CHECK(hipMalloc(&agentStartY, size_agent));
    CUDA_CHECK(hipMalloc(&agentDesX, size_agent));
    CUDA_CHECK(hipMalloc(&agentDesY, size_agent));
    CUDA_CHECK(hipMallocPitch(&agentWaypoints, &pitch, size_waypoint, size_agent));
    CUDA_CHECK(hipMalloc(&waypointX, size_agent));
    CUDA_CHECK(hipMalloc(&waypointY, size_agent));
    CUDA_CHECK(hipMalloc(&waypointR, size_agent));
    CUDA_CHECK(hipMalloc(&waypointIndex, size_agent));

    CUDA_CHECK(hipMemcpy(agentStartX, X.data(), size_agent, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(agentStartY, Y.data(), size_agent, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(agentDesX, X.data(), size_agent, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(agentDesY, Y.data(), size_agent, hipMemcpyHostToDevice));

    int *agentWaypointsTmp = new int[numAgents * numWaypoints];

    for (size_t i = 0; i < numAgents; i++)
    {
      for (size_t j = 0; j < numWaypoints; j++)
      {
        agentWaypointsTmp[i * numWaypoints + j] = agents.at(i)->getDestination().at(j)->getid();
      }
    }

    CUDA_CHECK(hipMemcpy2D(agentWaypoints, pitch, agentWaypointsTmp, size_waypoint, size_waypoint, numAgents,
                            hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(waypointX, X_WP.data(), size_waypoint, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(waypointY, Y_WP.data(), size_waypoint, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(waypointR, R_WP.data(), size_waypoint, hipMemcpyHostToDevice));

    int *waypointIndexTmp = new int[numAgents];

    for (size_t i = 0; i < numAgents; i++)
    {
      waypointIndexTmp[i] = agents.at(i)->getDestination().at(0)->getid();
    }

    CUDA_CHECK(hipMemcpy(waypointIndex, waypointIndexTmp, size_agent, hipMemcpyHostToDevice));

    tick_cuda(200, agentStartX, agentStartY, agentDesX, agentDesY, waypointX, waypointY, waypointR, agentWaypoints, pitch,
              waypointIndex);
  }
} // namespace Ped
