#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ped_model.h"
#include <>
#include <math.h>
#include <stdio.h>

//-----------------------------------------------------------------------------
// CUDA kernel to update agent positions.
// For each agent i, we do roughly the same as in sequential_tick():
//   diff = destination - current position
//   len = sqrt(diff.x*diff.x + diff.y*diff.y)
//   new position = current position + (diff/len)  (rounded)
// Then we store the new positions in desiredX/Y and update X/Y.
//-----------------------------------------------------------------------------
__global__ void cudaTickKernel(float *X, float *Y,
                               float *desiredX, float *desiredY,
                               const float *destX, const float *destY,
                               int numAgents)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numAgents)
    {
        float dx = destX[i] - X[i];
        float dy = destY[i] - Y[i];
        float len = sqrtf(dx * dx + dy * dy);

        if (len > 0.0f)
        {
            float newX = X[i] + dx / len;
            float newY = Y[i] + dy / len;

            desiredX[i] = roundf(newX);
            desiredY[i] = roundf(newY);
        }
        else
        {
            desiredX[i] = X[i];
            desiredY[i] = Y[i];
        }

        X[i] = desiredX[i];
        Y[i] = desiredY[i];
    }
}

namespace Ped
{
    void Ped::Model::cuda_tick()
    {
        int numAgents = agents.size();
        size_t size = numAgents * sizeof(float);

        for (int i = 0; i < numAgents; i++)
        {
            agents[i]->callNextDestination();
        }

        float *d_X, *d_Y, *d_desiredX, *d_desiredY, *d_destX, *d_destY;

        hipMalloc((void **)&d_X, size);
        hipMalloc((void **)&d_Y, size);
        hipMalloc((void **)&d_desiredX, size);
        hipMalloc((void **)&d_desiredY, size);
        hipMalloc((void **)&d_destX, size);
        hipMalloc((void **)&d_destY, size);

        hipMemcpy(d_X, Ped::X.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_Y, Ped::Y.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_desiredX, desiredX.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_desiredY, desiredY.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_destX, destinationX.data(), size, hipMemcpyHostToDevice);
        hipMemcpy(d_destY, destinationY.data(), size, hipMemcpyHostToDevice);

        int blockSize = 256;
        int gridSize = (numAgents + blockSize - 1) / blockSize;

        cudaTickKernel<<<gridSize, blockSize>>>(d_X, d_Y, d_desiredX, d_desiredY, d_destX, d_destY, numAgents);

        hipDeviceSynchronize();

        hipMemcpy(Ped::X.data(), d_X, size, hipMemcpyDeviceToHost);
        hipMemcpy(Ped::Y.data(), d_Y, size, hipMemcpyDeviceToHost);
        hipMemcpy(desiredX.data(), d_desiredX, size, hipMemcpyDeviceToHost);
        hipMemcpy(desiredY.data(), d_desiredY, size, hipMemcpyDeviceToHost);

        hipFree(d_X);
        hipFree(d_Y);
        hipFree(d_desiredX);
        hipFree(d_desiredY);
        hipFree(d_destX);
        hipFree(d_destY);

        for (int i = 0; i < numAgents; i++)
        {
            agents[i]->setX(desiredX[i]);
            agents[i]->setY(desiredY[i]);
        }
    }
}
